#include "hip/hip_runtime.h"
/*
 * main.cu
 *
 *  Created on: Sep 26, 2020
 *      Author: clark
 */

#include <stdio.h>
#include <stdlib.h>

#include <iostream>
#include <vector>
#include <thrust/device_vector.h>

#include "gtest/gtest.h"
#include "trajectory_data.cuh"
#include "g_tra_poptics.cuh"
#include "strtree.cuh"

/**
 * This macro checks return value of the CUDA runtime call and exits
 * the application if the call failed.
 */
#define CUDA_CHECK_RETURN(value) {											\
	hipError_t _m_cudaStat = value;										\
	if (_m_cudaStat != hipSuccess) {										\
		fprintf(stderr, "Error %s at line %d in file %s\n",					\
				hipGetErrorString(_m_cudaStat), __LINE__, __FILE__);		\
		exit(1);															\
	} }

/**
 *
 * Entry point for executing G-Tra-POPTICS
 */
int main(int argc, char **argv)
{
	std::string file_name = "data/testtrajectorydata.csv";

	// Load trajectory data from file
	file_trajectory_data trajectory_data = load_trajectory_data_from_file(file_name);

	// Preprocessing: build STR-tree index
	thrust::host_vector<strtree_line> lines = points_to_line_vector(trajectory_data.points, trajectory_data.trajectories,
			trajectory_data.num_points, trajectory_data.num_trajectories);

	// Build new trajectory index for vector "lines"
	thrust::host_vector<size_t> trajectory_start_indices(trajectory_data.num_trajectories);
	for (size_t i = 0, num_lines = 0; i < trajectory_data.num_trajectories; i++)
	{
		trajectory_start_indices[i] = num_lines;
		if (i < trajectory_data.num_trajectories - 1)
		{
			// Each trajectory has one fewer line than it had points.
			//num_lines += trajectory_data.trajectories[i + 1] - trajectory_data.trajectories[i] - 1;
		}
	}

	// Create index structure
	strtree strtree = cuda_create_strtree(lines);

	thrust::host_vector<strtree_offset_node> nodes = strtree.nodes;
	for(int i = 0; i < nodes.size(); i++)
	{
		strtree_offset_node node = nodes[i];
		std::cout << "Node " << i << ": num children=" << node.num << ", depth=" << node.depth << ", child_offset=" <<node.first_child_offset
			<< ", bbox.x1=" << node.boundingbox.x1 << ", bbox.x2=" << node.boundingbox.x2
			<< ", bbox.y1=" << node.boundingbox.y1 << ", bbox.y2=" << node.boundingbox.y2
			<< ", bbox.t1=" << node.boundingbox.t1 << ", bbox.t2=" << node.boundingbox.t2
			<< std::endl;
	}

//	/* Initialize variables for G-Tra-POPTICS execution */
//	// Number of CPU threads executing
//	int cpu_threads = 8;
//	// Maximum epsilon at which clusters are detected
//	double epsilon = 0.2;
//	// Specific epsilon for which to find clusters after minimum spanning trees are built
//	double epsilon_prime = 0.1;
//	// Minimum number of trajectories near a point for it to be considered a core point.
//	double min_num_trajectories = 2;
//
//	// Execute G-Tra-POPTICS on data file
//	g_tra_poptics(strtree, cpu_threads, epsilon, epsilon_prime, min_num_trajectories);

	/****** To run unit tests use this return ******/
	printf("Running unit tests from main.cu\n");
	testing::InitGoogleTest(&argc, argv);
	return RUN_ALL_TESTS();

	/****** To disable unit tests use this return ******/
//	return 0;
}



